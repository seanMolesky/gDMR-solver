#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <hip/hip_complex.h>
#include <hipblas.h>
#include "gDMR.h"
// GPU memory locations
hipComplex *srcLS;
hipComplex *imgLS;
hipComplex *optLS;
hipStream_t laStreamLS;
hipblasHandle_t laHandleLS;
hipError_t cuErrorLS;
const char *errorStringLS;
size_t matSizeLS;
size_t vecSizeLS;
int numEleLS;
int devNumLS;
// Internally accessible functions, device compatible.
__global__
void arrayInitLS(int rows, int cols, hipComplex *array)
{
	int localId = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;
	int matNum = rows * cols;
	int rowNum = 0;
	int colNum = 0;
	
	for (int i = localId; i < matNum; i += stride)
	{
		rowNum = i / rows;
		colNum = i % rows;

		if (rowNum == colNum)
		{
			array[i].x = 1.0;
			array[i].y = 0.0;
		}
		else if (colNum == rowNum + 1)
		{
			array[i].x = 0.0;
			array[i].y = 0.0;	
		}
		else
		{
			array[i].x = 0.0;
			array[i].y = 0.0;
		}
	}

	return;	
}


// Externally accessible functions.
extern "C"{__host__
	void resetLS(void)
	{
		if (hipSetDevice(0) != hipSuccess)
		{
			fprintf(stderr, "resetLS CUDA Error: Failed to reset LS device.\n");
			return;
		}			

		if (hipDeviceReset() != hipSuccess)
		{
			fprintf(stderr, "fullResetDMR CUDA Error: Failed to reset device.\n");
			return;
		}
		
		return;
	}
}

extern "C"{__host__
	void printCuErrorLS(const char *cuErrorString)
	{
		fprintf(stderr, "%s", cuErrorString);
		fprintf(stderr, ".\n");
		resetLS();
		return;
	}
}

extern "C"{__host__
	void testFuncLS(void)
	{	
		hipComplex prod;
		prod.x = 0.1;
		prod.y = 0.0;
		fprintf(stdout, "Test output not working %.2f.\n", sqrtf(prod.x));
		return;
	}
}

extern "C" {__host__
	void linOptLS(void)
	{	
		if (hipSetDevice(devNumLS) != hipSuccess)
		{
			fprintf(stderr, "optLS CUDA Error: Failed to set device.\n");
			resetLS();
			return;
		}

		hipComplex alpha, beta;
		alpha.x = 1.0;
		beta.x = 0.0;
		alpha.y = 0.0;
		beta.y = 0.0;

		if(hipblasCgemv(laHandleLS, HIPBLAS_OP_N, 
			numEleLS, numEleLS,
			&alpha,
			optLS, numEleLS,
			srcLS, 1,
			&beta,
			srcLS, 1) != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "optLS CUBLAS Error: Failed to preform matrix multiplication.\n");
			resetLS();
			return;	
		}

		return;
	}
}

extern "C"{__host__ 
	void initLS(int blocksDMR, int threadsPerBlockDMR, int devNumLO, int cells, int *devListDMR, int numDevsDMR, int basisSizeDMR, int deflatSizeDMR)
	{	
		numEleLS = cells;
		devNumLS = devNumLO;
		matSizeLS = sizeof(hipComplex) * numEleLS * numEleLS;
		vecSizeLS = sizeof(hipComplex) * numEleLS;

		if (hipSetDevice(devNumLO) != hipSuccess)
		{
			fprintf(stderr, "initLS CUDA Error: Failed to set device.\n");
			resetLS();
			return;
		}

		if (hipMallocManaged((void**) &optLS, matSizeLS) != hipSuccess) 
		{
			fprintf (stderr, "initLS CUDA Error: Failed to allocate device memory for global matrix.\n");
			resetLS();
			return;
		}

		if (hipMallocManaged((void**) &srcLS, vecSizeLS) != hipSuccess) 
		{
			fprintf(stderr, "initLS CUDA Error: Failed to allocate device memory for srcLS.\n");
			resetLS();
			return;
		}

		if (hipMallocManaged((void**) &imgLS, vecSizeLS) != hipSuccess) 
		{
			fprintf(stderr, "initLS CUDA Error: Failed to allocate device memory for imgLS.\n");
			resetLS();
			return;
		}

		if(hipStreamCreate(&laStreamLS) != hipSuccess)
		{
			fprintf(stderr, "initLS CUDA Error: Failed to initialize cublas stream.\n");
			resetLS();
			return;
		}

		if (hipblasCreate(&laHandleLS) != HIPBLAS_STATUS_SUCCESS) 
		{
			fprintf(stderr, "initLS CUBLAS Error: Failed to initialize cublas handle.\n");
			resetLS();
			return;
		}

		if(hipblasSetStream(laHandleLS, laStreamLS) != HIPBLAS_STATUS_SUCCESS) 
		{
			fprintf(stderr, "initLS CUBLAS Error: Failed to associate cublas handle and stream.\n");
			resetLS();
			return;	
		}

		arrayInitLS<<<blocksDMR, threadsPerBlockDMR, 0, laStreamLS>>>(numEleLS, numEleLS, optLS);

		initDMR(blocksDMR, threadsPerBlockDMR, devListDMR, numDevsDMR, numEleLS, basisSizeDMR, deflatSizeDMR, linOptLS);

		return;
	}
}

extern "C"{__host__
	void freeLS(void)
	{	
		if (hipSetDevice(devNumLS) != hipSuccess)
		{
			fprintf(stderr, "freeLS CUDA Error: Failed to set device.\n");
			resetLS();
			return;
		}

		if (hipFree(srcLS) != hipSuccess) 
		{
			fprintf(stderr, "freeLS CUDA Error: Failed to free device memory for srcLS.\n");
			resetLS();
			return;
		}

		if (hipFree(optLS) != hipSuccess) 
		{
			fprintf(stderr, "freeLS CUDA Error: Failed to free device memory for global matrix.\n");
			resetLS();
			return;
		}

		if (hipFree(imgLS) != hipSuccess) 
		{
			fprintf (stderr, "freeLS CUDA Error: Failed to free device memory for imgLS.\n");
			resetLS();
			return;
		}

		if (hipblasDestroy(laHandleLS) != HIPBLAS_STATUS_SUCCESS) 
		{
			fprintf (stderr, "freeLS CUBLAS Error: Failed to destroy cublas handle.\n");
			resetLS();
			return;
		}

		if(hipStreamDestroy(laStreamLS) != hipSuccess)
		{
			fprintf(stderr, "freeLS CUDA Error: Failed to destroy cublas stream.\n");
			resetLS();
			return;
		} 

		freeDMR();

		return;
	}
}

extern "C" {__host__
	void impImgLS(float _Complex *cImg)
	{
		if (hipSetDevice(devNumLS) != hipSuccess)
		{
			fprintf(stderr, "impImgLS CUDA Error: Failed to set device.\n");
			resetLS();
			return;
		}

		if(hipMemcpy(imgLS, cImg, vecSizeLS, hipMemcpyHostToDevice) != hipSuccess)
		{
			fprintf(stderr, "impImgLS CUDA Error: Failed to copy memory from host to device.\n");
			resetLS();
			return;
		}

		return;
	}
}

extern "C" {__host__
	void impSrcLS(float _Complex *cSrc)
	{	
		if (hipSetDevice(devNumLS) != hipSuccess)
		{
			fprintf(stderr, "impSrcLS CUDA Error: Failed to set device.\n");
			resetLS();
			return;
		}

		if(hipMemcpy(srcLS, cSrc, vecSizeLS, hipMemcpyHostToDevice) != hipSuccess)
		{
			fprintf(stderr, "impSrcLS CUDA Error: Failed to copy memory from host to device.\n");
			resetLS();
			return;
		}

		return;
	}
}

extern "C" {__host__
	void impOptLS(float _Complex *cOpt)
	{
		if (hipSetDevice(devNumLS) != hipSuccess)
		{
			fprintf(stderr, "impOptLS CUDA Error: Failed to set device.\n");
			resetLS();
			return;
		}

		if(hipMemcpy(optLS, cOpt, matSizeLS, hipMemcpyHostToDevice) != hipSuccess)
		{
			fprintf(stderr, "impOptLS CUDA Error: Failed to copy memory from host to device.\n");
			resetLS();
			return;
		}

		return;
	}
}

extern "C" {__host__
	void expImgLS(float _Complex *cImg)
	{	
		if (hipSetDevice(devNumLS) != hipSuccess)
		{
			fprintf(stderr, "expImgLS CUDA Error: Failed to set device.\n");
			resetLS();
			return;
		}

		cuErrorLS = hipMemcpy(cImg, imgLS, vecSizeLS, hipMemcpyDeviceToHost); 

		if(cuErrorLS != hipSuccess)
		{
			fprintf(stderr, "expImgLS CUDA Error: Failed to copy memory from device to host.\n");
			errorStringLS = hipGetErrorString(cuErrorLS);
			printCuErrorLS(errorStringLS);
			resetLS();
			return;
		}

		return;
	}
}

extern "C" {__host__
	void expSrcLS(float _Complex *cSrc)
	{	
		if (hipSetDevice(devNumLS) != hipSuccess)
		{
			fprintf(stderr, "expSrcLS CUDA Error: Failed to set device.\n");
			resetLS();
			return;
		}

		cuErrorLS = hipMemcpy(cSrc, srcLS, vecSizeLS, hipMemcpyDeviceToHost); 

		if(cuErrorLS != hipSuccess)
		{
			fprintf(stderr, " expSrcLS CUDA Error: Failed to copy memory from device to host.\n");
			errorStringLS = hipGetErrorString(cuErrorLS);
			printCuErrorLS(errorStringLS);
			resetLS();
			return;
		}

		return;
	}
}

extern "C" {__host__
	void expOptLS(float _Complex *cOpt)
	{	
		if (hipSetDevice(devNumLS) != hipSuccess)
		{
			fprintf(stderr, "expOptLS CUDA Error: Failed to set device.\n");
			resetLS();
			return;
		}

		cuErrorLS = hipMemcpy(cOpt, optLS, matSizeLS, hipMemcpyDeviceToHost); 

		if(cuErrorLS != hipSuccess)
		{
			fprintf(stderr, "expOptLS CUDA Error: Failed to copy memory from device to host.\n");
			errorStringLS = hipGetErrorString(cuErrorLS);
			printCuErrorLS(errorStringLS);
			resetLS();
			return;
		}

		return;
	}
}
// Solve linear system using gDMR, returning the residual norm of the solution.
// A mode value of !0 indicates that the solver has been previously called on a similar linear 
// system, and should use the existing deflation space in its first iteration.
extern "C" {__host__
	float solveLS(int mode, float solTol, int matRank, int *numIts)
	{
		return solverDMR(mode, srcLS, imgLS, laStreamLS, devNumLS, solTol, matRank, numIts);
	}
}