#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <hip/hip_complex.h>
#include <hipblas.h>
#include <hipsolver.h>
#include "gDMR.h"
// Arrays and coefficient vectors referencing the Krylov / deflation basis vectors. 
// hessArrDMR: Hessenberg coefficient array.
// hessSrcDMR: coefficients before multiplication with the Hessenberg matrix, 
// RHS, source side basis. 
// hessTrgDMR: coefficients after multiplication with the Hessenberg matrix,
// LHS, target side basis. 
// hessPrjDMR: projection coefficients / goal coefficients for a given iteration.
// hessWrkDMR: work space vector for reordering coefficient when multiple devices are 
// present.
hipComplex *hessArrDMR;
hipComplex *hessSrcDMR;
hipComplex *hessTrgDMR;
hipComplex *hessWrkDMR;
hipComplex *hessPrjDMR;
// Memory for QR decomposition of the Hessenberg matrix.
int wSizeQR;
hipComplex *qHessArrDMR;
hipComplex *rHessArrDMR;
// CUSOLVER arrays
hipComplex *tCUSLVDMR;
hipComplex *cpyHessArrDMR;
hipComplex *wCUSLVDMR;
hipsolverHandle_t slvHandleQRDMR;
hipStream_t slvStreamQRDMR;
// Memory for SV decompositions.
// uHessArrDMR: target side singular vectors.
// eHessArrDMR: singular values.
// vHessArrDMR: source side.
hipComplex *uHessArrDMR;
float *eHessArrDMR;
hipComplex *vHessArrDMR;
hipComplex *svWorkDMR;
hipsolverGesvdjInfo_t geSVjPrmsDMR;
hipsolverHandle_t slvHandleSVDMR;
hipStream_t slvStreamSVDMR;
int workSizeSVDMR;
// Additional memory for ``head" (master) device.
// genWorkVecDMR: general work space vector.
// sumWorkVecDMR: work space for partial sums.
// resVecDMR: image residual, persists across iterations.
// aSrcVecDMR: approximate source.
hipComplex *genWorkVecDMR;
hipComplex *sumWorkVecDMR;
hipComplex *resVecDMR;
hipComplex *aSrcVecDMR;
// Per device memory for generating vectors from the Krylov and deflation bases.
hipComplex **kryCoeffsDMR;
hipComplex **defCoeffsDMR;
// Per device memory holding basis vectors, and source side basis vectors for the deflation space.
hipComplex **itrBasisDMR;
hipComplex **defBasisDMR;
// Accompanying work areas.
hipComplex **itrVecDMR;
hipComplex **defWorkDMR;
// Inner product results.
hipComplex *innProdDevDMR;
// CUDA handles.
hipblasHandle_t *blasHandleDMR;
hipStream_t *blasStreamDMR;
hipblasStatus_t blasStatusDMR = HIPBLAS_STATUS_SUCCESS;
hipsolverStatus_t solverStatusDMR = HIPSOLVER_STATUS_SUCCESS;
// Global settings for GPUs
int *devListDMR;
int numDevsDMR;
int blocksDMR;
int threadsPerBlockDMR;
// Per device size of the dimension of total and deflation bases.
int localBasDimDMR;
int localDefDimDMR;
// Dimensionality of the total basis and deflation basis.
int basisDimDMR;
int deflateDimDMR;
// Number of elements in a solution vector.
int numElementsDMR;
size_t vecSizeDMR;
size_t hessenbergSizeDMR;
// SV decomposition settings
const hipsolverEigMode_t vecModeSVDMR = HIPSOLVER_EIG_MODE_VECTOR;
const int memModeSVDMR = 0;
const double svTolDMR = 2.e-7;
const int maxSVSweepsDMR = 128;
// Memory for function pointer providing linear iteration (matrix multiplication).
void (*linOptDMR)(void);
// Flag signifying that gDMR is stuck in a loop, and should move to set a different set
// of basis vectors.
int kryLoopFlagDMR;
int kryFlagCountDMR;
int kryBasisLoopsDMR;

__global__
void zeroArrayDMR(int startInd, int endInd, hipComplex *array)
{
	int localId = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;

	for(int i = localId + startInd; i < endInd; i += stride)
	{
		array[i].x = 0.0;
		array[i].y = 0.0;
	}

	return;		
}
// Reorder Hessenberg coefficient vector for consistency with multi device storage conventions.
// Mode -1 changes non-standard order to standard, all other integers standard to non-standard.
__global__
void reorderHessCoeffs(int mode, int numDevs, int startPos, int vecSize, int localBasisDim, const hipComplex *orgnVec, hipComplex *rordVec)
{
	int localId = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;
	int devOffset, memOffset;
	int rordPos;
	int j;

	if(mode != -1)
	{
		for(int i = localId; i < vecSize ; i += stride)
		{
			if(i < startPos)
			{
				rordVec[i] = orgnVec[i];
			}
			else
			{	
				j = i - startPos;
				devOffset = j % numDevs;
				memOffset = j / numDevs;
				rordPos = devOffset * localBasisDim + memOffset + startPos;
				rordVec[rordPos] = orgnVec[i];
			}
		}
	}
	else
	{
		for(int i = localId; i < vecSize ; i += stride)
		{
			if(i < startPos)
			{
				rordVec[i] = orgnVec[i];
			}
			else
			{
				j =  i - startPos;
				devOffset = j % numDevs;
				memOffset = j / numDevs;
				rordPos = devOffset * localBasisDim + memOffset + startPos;
				rordVec[i] = orgnVec[rordPos];
			}
		}
	}

	return;
}

__global__
void eyeArrayDMR(int startInd, int rows, int cols, hipComplex *array)
{
	int localId = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;
	int colNum, rowNum;

	for(int i = localId + startInd; i < rows * cols; i += stride)
	{
		rowNum = i % rows;
		colNum = i / rows;

		if(rowNum == colNum)
		{
			array[i].x = 1.0;
			array[i].y = 0.0;
		}
	}

	return;	
}

__global__
void vecAddDMR(int mode, int numElements, hipComplex *sumVec, const hipComplex *updateVec)
{
	int localId = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;

	if(mode == -1)
	{
		for(int i = localId; i < numElements; i += stride)
		{
			sumVec[i].x -= updateVec[i].x;
			sumVec[i].y -= updateVec[i].y;
		}
	}
	else
	{
		for(int i = localId; i < numElements; i += stride)
		{
			sumVec[i].x += updateVec[i].x;
			sumVec[i].y += updateVec[i].y;
		}	
	}

	return;		
}

__global__
void vecAddArrDMR(int arrayDim, int numElements, hipComplex *sumVec, const hipComplex *updateArray)
{
	int localId = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;

	for(int i = localId; i < numElements; i += stride)
	{
		for(int j = 0; j < arrayDim; j++)
		{
			sumVec[i].x += updateArray[i + j * numElements].x;
			sumVec[i].y += updateArray[i + j * numElements].y;
		}
	}

	return;
}

__global__
void vecScaleDiffDMR(int numCoeffs, int numCells, const hipComplex *coeffs, const hipComplex *vecBasis, float scale, hipComplex *mutateVec)
{
	int localId = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;

	for(int i = localId; i < numCells; i += stride)
	{	
		mutateVec[i].x = mutateVec[i].x * scale;
		mutateVec[i].y = mutateVec[i].y * scale;

		for(int j = 0; j < numCoeffs; j++)
		{
			mutateVec[i].x -= coeffs[j].x * vecBasis[j * numCells + i].x - coeffs[j].y * vecBasis[j * numCells + i].y;
			mutateVec[i].y -= coeffs[j].x * vecBasis[j * numCells + i].y + coeffs[j].y * vecBasis[j * numCells + i].x;
		}
	}

	return;
}

__global__
void addBasisVecsDMR(int numCoeffs, int numCells, const hipComplex *coeffs, const hipComplex *vecBasis, hipComplex *mutateVec)
{
	int localId = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;

	for(int i = localId; i < numCells; i += stride)
	{	
		for(int j = 0; j < numCoeffs; j++)
		{
			mutateVec[i].x += coeffs[j].x * vecBasis[j * numCells + i].x - coeffs[j].y * vecBasis[j * numCells + i].y;
			mutateVec[i].y += coeffs[j].x * vecBasis[j * numCells + i].y + coeffs[j].y * vecBasis[j * numCells + i].x;
		}
	}

	return;
}

__global__
void innerProdKerDMR(int numElements, int blocks, const hipComplex *linFunc, const hipComplex *vec, hipComplex *blockProds, hipComplex *prod)
{	
	extern __shared__ hipComplex lCache[];

	hipComplex threadSum;
	threadSum.x = 0.0;
	threadSum.y = 0.0;
	int lId = threadIdx.x;
	int gId = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;
	int rLevel = blockDim.x / 2;

	for(int i = gId; i < numElements; i += stride)
	{
		threadSum.x += linFunc[i].x * vec[i].x + linFunc[i].y * vec[i].y;
		threadSum.y += linFunc[i].x * vec[i].y - linFunc[i].y * vec[i].x;
	}

	lCache[lId].x = threadSum.x;
	lCache[lId].y = threadSum.y;
	__syncthreads();

	while (rLevel != 0)
	{
		if(lId < rLevel)
		{
			lCache[lId].x += lCache[lId+rLevel].x;
			lCache[lId].y += lCache[lId+rLevel].y;
		}
		__syncthreads();
		rLevel /= 2;
	}

	if(lId == 0)
	{
		blockProds[blockIdx.x].x = lCache[0].x;
		blockProds[blockIdx.x].y = lCache[0].y;
	}
	
	__syncthreads();

	if(gId == 0)
	{
		prod[0].x = 0.0;
		prod[0].y = 0.0;

		for(int i = 0; i < blocks; i++)
		{
			prod[0].x += blockProds[i].x;
			prod[0].y += blockProds[i].y;
		}

		prod[0].x = prod[0].x;
		prod[0].y = prod[0].y;
	}

	return;	
}

__global__ 
void normalizeVecDMR(int mode, int numCells, hipComplex *vec, hipComplex *norm, float nNorm)
{ 
	int localId = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;
	float realNorm;

	if(mode == -1)
	{
		realNorm = 1.0 / sqrtf(norm[0].x);
	}
	else if(mode == 2)
	{
		realNorm = nNorm;
	}
	else
	{
		realNorm = sqrtf(norm[0].x);
	}
	
	for(int i = localId; i < numCells; i += stride)
	{
		vec[i].x = vec[i].x * realNorm;
		vec[i].y = vec[i].y * realNorm;
	}

	return;	
}
// Function for examining device data. Otherwise unessential.
extern "C"{__host__
	void gPrintDMR(int rows, int cols, hipComplex *gArray, int devNum)
	{
		int rowNum;
		int colNum;

		if(hipSetDevice(devNum) != hipSuccess)
		{
			fprintf(stderr, "gPrintDMR CUDA Error: Failed to switch to device %d.\n", devNum);
			return;
		}

		if(hipDeviceSynchronize() != hipSuccess)
		{
			fprintf(stderr, "gPrintDMR CUDA Error: Failed to synchronize device %d.\n", devNum);
			return;	
		}

		hipComplex *hostMem;
		hostMem = (hipComplex*)malloc(sizeof(hipComplex) * rows * cols);

		if(hipMemcpy(hostMem, gArray, sizeof(hipComplex) * rows * cols, hipMemcpyDeviceToHost) != hipSuccess)
		{
			fprintf(stderr, "gPrintDMR CUDA Error: Failed to copy data to host.\n");
			return;	
		}	

		fprintf(stdout, "\n");

		for(int itr = 0; itr < rows * cols; itr++)
		{
			colNum = itr % cols;
			rowNum = itr / cols;

			if((itr + 1) % cols == 0)
			{
				fprintf(stdout, "%4.3f+%4.3fi\n", hostMem[rowNum + colNum * rows].x, hostMem[rowNum + colNum * rows].y);
			}
			else
			{
				fprintf(stdout, "%4.3f+%4.3fi ", hostMem[rowNum + colNum * rows].x, hostMem[rowNum + colNum * rows].y);
			}
		}

		fprintf(stdout, "\n");
		free(hostMem);
		return;
	}
}
// Resets all device memory, called if an error is detected. Note that this function will 
// produce segmentation faults if called for multiple virtual devices residing on the same 
// physical device.
extern "C"{__host__
	void fullResetDMR(int devUpperBound)
	{
		for(int devItr = 0; devItr < devUpperBound; devItr++)
		{
			if(hipSetDevice(devListDMR[devItr]) != hipSuccess)
			{
				fprintf(stderr, "fullResetDMR CUDA Error: Failed to switch to device %d.\n", devItr);
				return;
			}			

			if(hipDeviceReset() != hipSuccess)
			{
				fprintf(stderr, "fullResetDMR CUDA Error: Failed to reset device %d.\n", devItr);
				return;
			}
		}		

		return;
	}
}

extern "C"{__host__
	void impSVDMR(float _Complex *cMem)
	{
		for(int devItr = 0; devItr < numDevsDMR; devItr++)
		{	
			if (hipSetDevice(devItr) != hipSuccess)
			{
				fprintf(stderr, "impSVDMR CUDA Error: Failed to set device.\n");
				resetLS();
				return;
			}
			
			if(hipMemcpy(defBasisDMR[devItr], &(cMem[devItr * numElementsDMR * localDefDim]), sizeof(hipComplex) * numElementsDMR * localDefDim, hipMemcpyHostToDevice) != hipSuccess)
			{
				fprintf(stderr, "impSVDMR CUDA Error: Failed to import singular vectors on device %d.\n", devItr);
				resetLS();
				return;
			}
		}

		return;
	}
}

extern "C"{__host__
	void expSVDMR(float _Complex *cMem)
	{
		for(int devItr = 0; devItr < numDevsDMR; devItr++)
		{	
			if (hipSetDevice(devItr) != hipSuccess)
			{
				fprintf(stderr, "impSVDMR CUDA Error: Failed to set device.\n");
				resetLS();
				return;
			}
			
			if(hipMemcpy(&(cMem[devItr * numElementsDMR * localDefDim]), defBasisDMR[devItr], sizeof(hipComplex) * numElementsDMR * localDefDim, hipMemcpyHostToDevice) != hipSuccess)
			{
				fprintf(stderr, "impSVDMR CUDA Error: Failed to export singular vectors from device %d.\n", devItr);
				resetLS();
				return;
			}
		}

		return;
	}
}

extern "C"{__host__
	int isPowerTwo(int n) 
	{ 
		if(fmod(log2((float) n), 1.0) < 0.01)
		{
			return 0;
		}
		else
		{
			return 1;
		} 
	}
}

extern "C"{__host__
	int innerProdDMR(int numElements, const hipComplex *linFunc, const hipComplex *vec, hipComplex *prod, int devNum)
	{
		hipStream_t prodStream;
		hipComplex *blockProds;

		if(hipSetDevice(devNum) != hipSuccess)
		{
			fprintf(stderr, "innerProdDMR CUDA Error: Failed to switch to device %d.\n", devNum);
			fullResetDMR(numDevsDMR);
			return 1;
		}

		if(isPowerTwo(blocksDMR) != 0)
		{
			fprintf(stderr, "innerProdDMR Error: Reduction algorithm requires the number of blocksDMR, %d, to be a power of 2.\n", blocksDMR);
			fullResetDMR(numDevsDMR);
			return 1;
		}

		if(hipMalloc((void**) &blockProds, blocksDMR * sizeof(hipComplex)) != hipSuccess) 
		{
			fprintf(stderr, "innerProdDMR CUDA Error: Failed to allocate device memory for product reduction.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		if(hipStreamCreate(&prodStream) != hipSuccess)
		{
			fprintf(stderr, "innerProdDMR CUDA Error: Failed to initialize stream for inner product.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		innerProdKerDMR<<<blocksDMR, threadsPerBlockDMR, threadsPerBlockDMR * sizeof(hipComplex), prodStream>>>(numElements, blocksDMR, linFunc, vec, blockProds, prod);

		if(hipStreamSynchronize(prodStream) != hipSuccess) 
		{
			fprintf(stderr, "innerProdDMR CUDA Error: Failed to synchronize inner product stream.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		hipFree(blockProds);

		if(hipStreamDestroy(prodStream) != hipSuccess)
		{
			fprintf(stderr, "innerProdDMR CUDA Error: Failed to destroy inner product stream.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		return 0;
	}
}
// Computes projection coefficients of a source vector against the column vectors of 
// a matrix, operationMat. The result is stored in target.
extern "C"{__host__
	int basisProjDMR(const hipComplex* operationMat, const hipComplex* source, hipComplex* target, int basisDim, int numElements, int devItr)
	{
		hipComplex alpha, beta;
		alpha.x = 1.0;
		alpha.y = 0.0;
		beta.x = 0.0;
		beta.y = 0.0;

		if(hipSetDevice(devListDMR[devItr]) != hipSuccess)
		{
			fprintf(stderr, "basisProjDMR CUDA Error: Failed to switch to device %d.\n", devListDMR[devItr]);
			fullResetDMR(numDevsDMR);
			return 1;
		}

		blasStatusDMR = hipblasCgemv(blasHandleDMR[devItr], HIPBLAS_OP_C,
			numElements, basisDim,
			&alpha,
			operationMat, numElements,
			source, 1,
			&beta,
			target, 1);

		if(hipStreamSynchronize(blasStreamDMR[devItr]) != hipSuccess)
		{
			fprintf(stderr, "basisProjDMR CUDA Error: Failed to synchronize BLAS stream %d.\n", devItr);
			fullResetDMR(numDevsDMR);
			return 1;
		}

		if(blasStatusDMR != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "basisProjDMR CUBLAS Error: Basis projection has failed.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		return 0;
	}
}
// Calculates the linear action of the Hessenberg matrix on the coefficients in hessSrcDMR.
// Result is stored in hessTrgDMR.
extern "C"{__host__
	int hessenbergMultDMR(void)
	{
		hipComplex alpha, beta;
		alpha.x = 1.0;
		alpha.y = 0.0;
		beta.x = 0.0;
		beta.y = 0.0;

		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "hessenbergMultDMR CUDA Error: Failed to set device to head solver.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		blasStatusDMR = hipblasCgemv(blasHandleDMR[0], HIPBLAS_OP_N,
			basisDimDMR + 1, basisDimDMR,
			&alpha,
			hessArrDMR, basisDimDMR + 1,
			hessSrcDMR, 1,
			&beta,
			hessWrkDMR, 1);

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "hessenbergMultDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		if(blasStatusDMR != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "hessenbergMultDMR CUBLAS Error: Failed to apply Hessenberg operation.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		return 0;
	}
}
// Performs SV decomposition factorization of a source array, sArray.
// Results for the target singular vectors are stored in uArray, source singular vectors vArray,
// and singular values eArray. 
extern "C"{__host__
	int svDecompHessDMR(void)
	{
		int info = 0;
		int *devInfo = NULL;

		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "svDecompHessDMR CUDA Error: Failed to set device to head solver.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		if(hipMalloc((void**) &devInfo, sizeof(int)) != hipSuccess) 
		{
			fprintf(stderr, "svDecompHessDMR CUDA Error: Failed to allocate device memory for SV information.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}
		
		solverStatusDMR = hipsolverDnCgesvdj(slvHandleSVDMR, vecModeSVDMR, memModeSVDMR,
			basisDimDMR + 1, basisDimDMR,
			hessArrDMR, basisDimDMR + 1,
			eHessArrDMR,
			uHessArrDMR, basisDimDMR + 1,
			vHessArrDMR, basisDimDMR,
			svWorkDMR, workSizeSVDMR,
			devInfo, geSVjPrmsDMR);

		if(hipStreamSynchronize(slvStreamSVDMR) != hipSuccess)
		{
			fprintf(stderr, "svDecompHessDMR CUDA Error: Failed to synchronize solver stream.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		if(solverStatusDMR != HIPSOLVER_STATUS_SUCCESS)
		{
			if(hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
			{
				fprintf(stderr, "svDecompHessDMR CUDA Error: Failed to copy SV information to host.\n");
				return 1;	
			}
			
			fprintf(stderr, "svDecompHessDMR CUDASOLVER Error: SV failed with error %d.\n", info);
			fullResetDMR(numDevsDMR);
			return 1;	
		}
		// Free device memory 
		hipFree(devInfo);
		return 0;
	}
}
// Performs QR decomposition of Hessenberg matrix.
extern "C"{__host__
	int qrHessArrDMR(void)
	{
		int solverInt = 0;
		int *devInfo = NULL;

		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "qrHessArrDMR CUDA Error: Failed to set device to head solver.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		if(hipMallocManaged((void**) &devInfo, sizeof(int)) != hipSuccess) 
		{
			fprintf(stderr, "qrHessArrDMR CUDA Error: Failed to allocate device memory for device response.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		if(hipMemcpyPeerAsync(cpyHessArrDMR, devListDMR[0], hessArrDMR, devListDMR[0], hessenbergSizeDMR, slvStreamQRDMR) != hipSuccess)
		{
			fprintf(stderr, "qrHessArrDMR CUDA Error: Failed to copy Hessenberg array.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}
		// Perform QR factorization in two steps.
		// First step of QR
		solverStatusDMR = hipsolverDnCgeqrf(slvHandleQRDMR, 
			basisDimDMR + 1, basisDimDMR, 
			cpyHessArrDMR, basisDimDMR + 1, 
			tCUSLVDMR, 
			wCUSLVDMR, wSizeQR, 
			devInfo);

		if(hipStreamSynchronize(slvStreamQRDMR) != hipSuccess)
		{
			fprintf(stderr, "qrHessArrDMR CUDA Error: Failed to synchronize solver stream.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		if(solverStatusDMR != HIPSOLVER_STATUS_SUCCESS)
		{
			fprintf(stderr, "qrHessArrDMR CUSOLVER Error: Failed first step of QR factorization.\n");
			hipMemcpy(&solverInt, devInfo, sizeof(int), hipMemcpyDeviceToHost);
			fprintf(stdout, "Geqrf integer: %d.\n", solverInt);
			fullResetDMR(numDevsDMR);
			return 1;	
		}
		// Export the R array. Note that R is completely filled, but only the right hand
		// (upper) part of the matrix contains useful data. This is consistent with the 
		// convention of cuda solvers for triangular matrices (toolkit v10.0.13).
		for(int colItr = 0; colItr < basisDimDMR; colItr++)
		{
			if(hipMemcpyPeerAsync(&(rHessArrDMR[basisDimDMR * colItr]), devListDMR[0], &(cpyHessArrDMR[(basisDimDMR + 1) * colItr]), devListDMR[0], sizeof(hipComplex) * basisDimDMR, slvStreamQRDMR) != hipSuccess)
			{
				fprintf(stderr, "qrHessArrDMR CUDA Error: Failed to copy column %d of upper triangular decomposition of Hessenberg array.\n", colItr);
				fullResetDMR(numDevsDMR);
				return 1;
			}
		}

		if(hipStreamSynchronize(slvStreamQRDMR) != hipSuccess)
		{
			fprintf(stderr, "qrHessArrDMR CUDA Error: Failed to synchronize solver stream.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}
		// Second step of QR
		solverStatusDMR = hipsolverDnCungqr(slvHandleQRDMR,
			basisDimDMR + 1, basisDimDMR, basisDimDMR,
			cpyHessArrDMR, basisDimDMR + 1,
			tCUSLVDMR,
			wCUSLVDMR, wSizeQR,
			devInfo);

		if(hipStreamSynchronize(slvStreamQRDMR) != hipSuccess)
		{
			fprintf(stderr, "qrHessArrDMR CUDA Error: Failed to synchronize solver stream.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		if(solverStatusDMR != HIPSOLVER_STATUS_SUCCESS)
		{
			fprintf(stderr, "qrHessArrDMR CUSOLVER Error: Failed second step of QR factorization.\n");
			hipMemcpy(&solverInt, devInfo, sizeof(int), hipMemcpyDeviceToHost);
			fprintf(stdout, "Ungqr integer: %d.\n", solverInt);
			fullResetDMR(numDevsDMR);
			return 1;	
		}

		if(hipMemcpyPeerAsync(qHessArrDMR, devListDMR[0], cpyHessArrDMR, devListDMR[0], hessenbergSizeDMR, slvStreamQRDMR) != hipSuccess)
		{
			fprintf(stderr, "qrHessArrDMR CUDA Error: Failed to copy Q basis of Hessenberg array decomposition.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		if(hipStreamSynchronize(slvStreamQRDMR) != hipSuccess)
		{
			fprintf(stderr, "qrHessArrDMR CUDA Error: Failed to synchronize solver stream.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}
		// Free temporary memory
		hipFree(devInfo);   
		return 0;
	}
}
// Solves A x = s given a qr factorization of A.
extern "C"{__host__
	int qrSolveDMR(int rows, int cols, hipComplex *trgVec, hipComplex *srcVec, const hipComplex *qArray, hipComplex *rArray)
	{
		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "qrSolveDMR CUDA Error: Failed to set device to head solver.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		if(basisProjDMR(qArray, trgVec, srcVec, cols, rows, devListDMR[0]) != 0)
		{
			return 1;
		}

		blasStatusDMR = hipblasCtrsv(blasHandleDMR[0], HIPBLAS_FILL_MODE_UPPER,
			HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
			cols, rArray, cols,
			srcVec, 1);

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "qrSolveDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		if(blasStatusDMR != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "qrSolveDMR CUBLAS Error: Failure in QR solver.\n");
			fullResetDMR(numDevsDMR);
			return 1;	
		}

		return 0;
	}
}
// Initialize global device memory.
extern "C"{__host__
	void initDMR(int blocks, int threadsPerBlock, int *devList, int numDevs, int elements, int basisDim, int deflateDim, void(*linOpt)(void))
	{	
		if((basisDim + 1) % numDevs != 0)
		{
			fprintf(stderr, "initDMR Error: The dimension basisDim + 1 must be divisible by the number of requested devices.\n");
			return;
		}

		if(deflateDim % numDevs != 0) 
		{
			fprintf(stderr, "initDMR Error: The deflation space dimension must be divisible by the number of requested devices.\n");
			return;
		}

		if(basisDim - deflateDim < 2)
		{
			fprintf(stderr, "initDMR Error: After removing the deflation space, the basis dimension is too small to perform Arnoldi iterations.\n");
			return;
		}

		int wSizeQR1 = 0, wSizeQR2 = 0;
		linOptDMR = linOpt;
		blocksDMR = blocks;
		numElementsDMR = elements;
		threadsPerBlockDMR = threadsPerBlock;
		// Basis settings.
		numDevsDMR = numDevs;
		basisDimDMR = basisDim;
		deflateDimDMR = deflateDim;
		localBasDimDMR = (basisDim + 1) / numDevs;
		localDefDimDMR = deflateDim / numDevs;
		// Memory sizes.
		vecSizeDMR = sizeof(hipComplex) * elements;
		hessenbergSizeDMR = sizeof(hipComplex) * (basisDimDMR + 1) * basisDimDMR;
		size_t matSize = sizeof(hipComplex) * basisDimDMR * basisDimDMR;
		devListDMR = (int*)malloc(sizeof(int) * numDevs);
		// Per device memory.
		defCoeffsDMR = (hipComplex**)malloc(sizeof(hipComplex*) * numDevs);
		kryCoeffsDMR = (hipComplex**)malloc(sizeof(hipComplex*) * numDevs);
		itrBasisDMR = (hipComplex**)malloc(sizeof(hipComplex*) * numDevs);
		itrVecDMR = (hipComplex**)malloc(sizeof(hipComplex*) * numDevs);
		defBasisDMR = (hipComplex**)malloc(sizeof(hipComplex*) * numDevs);
		defWorkDMR = (hipComplex**)malloc(sizeof(hipComplex*) * numDevs);
		// Handles
		blasHandleDMR = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t) * numDevs);
		blasStreamDMR = (hipStream_t*)malloc(sizeof(hipStream_t) * numDevs); 
		// Krylov loop flag parameters
		kryLoopFlagDMR = 0;
		kryFlagCountDMR = 0;
		kryBasisLoopsDMR = 0;

		for(int devItr = 0; devItr < numDevs; devItr++)
		{	
			devListDMR[devItr] = devList[devItr];

			if(hipSetDevice(devListDMR[devItr]) != hipSuccess)
			{
				fprintf(stderr, "initDMR CUDA Error: Failed to set device to %d.\n", devItr);
				return;
			}

			if(devItr == 0)
			{
				if(hipMallocManaged((void**) &hessArrDMR, hessenbergSizeDMR) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate device memory for Hessenberg matrix on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipMallocManaged((void**) &genWorkVecDMR, vecSizeDMR) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate general work memory on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(numDevsDMR > 1)
				{
					if(hipMallocManaged((void**) &sumWorkVecDMR, (numDevsDMR - 1) * vecSizeDMR) != hipSuccess) 
					{
						fprintf(stderr, "initDMR CUDA Error: Failed to allocate device memory for summing iteration vectors on head solver.\n");
						fullResetDMR(devItr);
						return;
					}
				}

				if(hipMallocManaged((void**) &resVecDMR, vecSizeDMR) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate device memory for residual vector on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipMallocManaged((void**) &aSrcVecDMR, vecSizeDMR) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate device memory for approximate solution on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipMallocManaged((void**) &qHessArrDMR, hessenbergSizeDMR) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate device memory for basis vectors of Hessenberg QR decomposition on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipMallocManaged((void**) &rHessArrDMR, matSize) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate device memory for upper matrix of Hessenberg QR decomposition on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipMallocManaged((void**) &hessSrcDMR, sizeof(hipComplex) * basisDimDMR) != hipSuccess) 
				{
					fprintf (stderr, "initDMR CUDA Error: Failed to allocate device memory for Hessenberg source side work basis coefficients on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipMallocManaged((void**) &hessTrgDMR, sizeof(hipComplex) * (basisDimDMR + 1)) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate device memory for Hessenberg target side basis coefficients on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipMallocManaged((void**) &hessWrkDMR, sizeof(hipComplex) * (basisDimDMR + 1)) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate device memory for Hessenberg target side work basis coefficients on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipMallocManaged((void**) &hessPrjDMR, sizeof(hipComplex) * (basisDimDMR + 1)) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate device memory for source basis coefficients on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipMallocManaged((void**) &uHessArrDMR, hessenbergSizeDMR) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate device memory for target basis of SV decomposition on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipMallocManaged((void**) &eHessArrDMR, sizeof(float) * basisDimDMR) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate device memory for singular values of Hessenberg on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipMallocManaged((void**) &vHessArrDMR, matSize) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate device memory for source space of SV decomposition on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipMalloc((void**) &innProdDevDMR, sizeof(hipComplex)) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate device memory for inner products on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}
				// Initialize solver handle and streams for SV decomposition.
				if(hipsolverDnCreate(&slvHandleSVDMR) != HIPSOLVER_STATUS_SUCCESS)
				{
					fprintf(stderr, "initDMR CUSOLVER Error: Failed to create solver handle for SV decomposition on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipStreamCreateWithFlags(&slvStreamSVDMR, hipStreamNonBlocking) != 
					hipSuccess)
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to initialize cuSOLVER stream for SV decomposition on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipsolverSetStream(slvHandleSVDMR, slvStreamSVDMR) != HIPSOLVER_STATUS_SUCCESS)
				{
					fprintf(stderr, "initDMR CUSOLVER Error: Failed to set cuSOLVER handle to stream for SV decomposition on head solver.\n");
					fullResetDMR(devItr);
					return;
				}
				// Configure SV
				if(hipsolverDnCreateGesvdjInfo(&geSVjPrmsDMR) != HIPSOLVER_STATUS_SUCCESS)
				{
					fprintf(stderr, "initDMR CUSOLVER Error: Failed to set SV decomposition configuration memory location on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipsolverDnXgesvdjSetTolerance(geSVjPrmsDMR, svTolDMR) != HIPSOLVER_STATUS_SUCCESS)
				{
					fprintf(stderr, "initDMR CUSOLVER Error: Failed to set SV decomposition tolerance on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipsolverDnXgesvdjSetMaxSweeps(geSVjPrmsDMR, maxSVSweepsDMR) != HIPSOLVER_STATUS_SUCCESS)
				{
					fprintf(stderr, "initDMR CUSOLVER Error: Failed to set SV decomposition max sweeps on head solver.\n");
					fullResetDMR(devItr);
					return;	
				} 
				// Determine workspace required for SV
				if(hipsolverDnCgesvdj_bufferSize(slvHandleSVDMR, vecModeSVDMR, memModeSVDMR,
					basisDimDMR + 1, basisDimDMR,
					hessArrDMR, basisDimDMR + 1,
					eHessArrDMR,
					uHessArrDMR, basisDimDMR + 1,
					vHessArrDMR, basisDimDMR,
					&workSizeSVDMR, geSVjPrmsDMR) != HIPSOLVER_STATUS_SUCCESS)
				{
					fprintf(stderr, "initDMR CUSOLVER Error: Failed to determine SV decomposition work size on head solver.\n");
					fullResetDMR(devItr);
					return;		
				}
				// Allocate SV workspace
				if(hipMallocManaged((void**) &svWorkDMR, sizeof(hipComplex) * workSizeSVDMR) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate device workspace for SV decomposition on head solver.\n");
					fullResetDMR(devItr);
					return;
				}
				// Initialize solver handle and streams for QR factorization.
				if(hipsolverDnCreate(&slvHandleQRDMR) != HIPSOLVER_STATUS_SUCCESS)
				{
					fprintf(stderr, "initDMR CUSOLVER Error: Failed to create solver handle for QR decomposition on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipStreamCreate(&slvStreamQRDMR) != hipSuccess)
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to initialize cuSOLVER stream for QR decomposition on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipsolverSetStream(slvHandleQRDMR, slvStreamQRDMR) != HIPSOLVER_STATUS_SUCCESS)
				{
					fprintf(stderr, "initDMR CUSOLVER Error: Failed to set cuSOLVER handle to stream for QR decomposition on head solver.\n");
					fullResetDMR(devItr);
					return;		
				}	
				// Initialize local memory for QR solver.
				if(hipMallocManaged((void**) &tCUSLVDMR, sizeof(hipComplex) * basisDimDMR) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate device memory for Tau array (QR decomposition) on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipMallocManaged((void**) &cpyHessArrDMR, hessenbergSizeDMR) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate device memory QR array copy on head solver.\n");
					fullResetDMR(devItr);
					return;
				}
				// Calculate required size of the work areas
				if(hipsolverDnCgeqrf_bufferSize(slvHandleQRDMR,
					basisDimDMR + 1, basisDimDMR,
					cpyHessArrDMR, basisDimDMR + 1, &wSizeQR1) != HIPSOLVER_STATUS_SUCCESS)
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to determine work area requirements for first step of QR decomposition on head solver.\n");
					fullResetDMR(devItr);
					return;
				}

				if(hipsolverDnCungqr_bufferSize(slvHandleQRDMR,
					basisDimDMR + 1, basisDimDMR, basisDimDMR,
					cpyHessArrDMR, basisDimDMR + 1,
					tCUSLVDMR, &wSizeQR2) != HIPSOLVER_STATUS_SUCCESS)
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to determine work area requirements for second step of QR decomposition on head solver.\n");
					fullResetDMR(devItr);
					return;
				}
		    	// Set size of work area to maximum required size. 
				wSizeQR = (wSizeQR1 > wSizeQR2) ? wSizeQR1 : wSizeQR2;

				if(hipMallocManaged((void**) &wCUSLVDMR, sizeof(hipComplex) * wSizeQR) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate work area for QR factorization on head solver.\n");
					fullResetDMR(devItr);
					return;
				}
			}

			if(hipStreamCreate(&(blasStreamDMR[devItr])) != hipSuccess)
			{
				fprintf(stderr, "initDMR CUDA Error: Failed to initialize cuBLAS stream on device %d.\n", devItr);
				fullResetDMR(devItr);
				return;
			}

			if(hipblasCreate(&(blasHandleDMR[devItr])) != HIPBLAS_STATUS_SUCCESS) 
			{
				fprintf(stderr, "initDMR CUBLAS Error: Failed to initialize cuBLAS handle on device %d.\n", devItr);
				fullResetDMR(devItr);
				return;
			}

			if(hipblasSetStream(blasHandleDMR[devItr], blasStreamDMR[devItr]) != HIPBLAS_STATUS_SUCCESS) 
			{
				fprintf(stderr, "initDMR CUBLAS Error: Failed to set cuBLAS handle to cuBLAS stream on device %d.\n", devItr);
				fullResetDMR(devItr);
				return;	
			}

			if(hipMallocManaged((void**) &(itrBasisDMR[devItr]), vecSizeDMR * localBasDimDMR) != hipSuccess) 
			{
				fprintf(stderr, "initDMR CUDA Error: Failed to allocate memory for Krylov basis on device %d.\n", devItr);
				fullResetDMR(devItr);
				return;
			}

			if(localDefDimDMR > 0)
			{
				if(hipMallocManaged((void**) &(defBasisDMR[devItr]), vecSizeDMR * localDefDimDMR) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate memory for source side singular vectors on device %d.\n", devItr);
					fullResetDMR(devItr);
					return;
				}

				if(hipMallocManaged((void**) &(defWorkDMR[devItr]), vecSizeDMR * localDefDimDMR) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate work area for source side singular vectors on device %d.\n", devItr);
					fullResetDMR(devItr);
					return;
				}

				if(hipMallocManaged((void**) &(defCoeffsDMR[devItr]), sizeof(hipComplex) * localDefDimDMR) != hipSuccess) 
				{
					fprintf(stderr, "initDMR CUDA Error: Failed to allocate memory for deflation coefficients on device %d.\n", devItr);
					fullResetDMR(devItr);
					return;
				}
			}

			if(hipMallocManaged((void**) &(kryCoeffsDMR[devItr]), sizeof(hipComplex) * localBasDimDMR) != hipSuccess) 
			{
				fprintf(stderr, "initDMR CUDA Error: Failed to allocate memory for Krylov basis coefficients on device %d.\n", devItr);
				fullResetDMR(devItr);
				return;
			}

			if(hipMallocManaged((void**) &(itrVecDMR[devItr]), vecSizeDMR) != hipSuccess) 
			{
				fprintf(stderr, "initDMR CUDA Error: Failed to allocate memory for temporary solutions on device %d.\n", devItr);
				fullResetDMR(devItr);
				return;
			}
		}
		return;
	}
}

extern "C"{__host__
	void freeDMR(void)
	{	
		for(int devItr = 0; devItr < numDevsDMR; devItr++)
		{	

			if(hipSetDevice(devListDMR[devItr]) != hipSuccess)
			{
				fprintf(stderr, "freeDMR CUDA Error: Failed to set active device to %d.\n", devItr);
				fullResetDMR(numDevsDMR);
				return;
			}

			if(devItr == 0)
			{
				if(hipFree(hessArrDMR) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear memory for Hessenberg matrix on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree(genWorkVecDMR) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear general work area on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(numDevsDMR > 1)
				{
					if(hipFree(sumWorkVecDMR) != hipSuccess) 
					{
						fprintf(stderr, "freeDMR CUDA Error: Failed to clear sum work area on head solver.\n");
						fullResetDMR(numDevsDMR);
						return;
					}
				}

				if(hipFree(resVecDMR) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear memory for residual vector on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree(aSrcVecDMR) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear memory for the approximate solution on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree(qHessArrDMR) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear memory for vectors of QR Hessenberg decomposition on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree(rHessArrDMR) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear memory for upper triangular matrix of Hessenberg decomposition on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree(hessSrcDMR) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear memory for source side Hessenberg work coefficients on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree(hessTrgDMR) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear memory for Hessenberg target side coefficients on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree(hessWrkDMR) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear memory for Hessenberg target side work coefficients on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree(hessPrjDMR) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear memory for source basis coefficients on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree(uHessArrDMR) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear memory for SV target basis on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree(eHessArrDMR) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear memory for singular values on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree(vHessArrDMR) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear memory for input singular values on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree(svWorkDMR) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear device SV work area on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree(innProdDevDMR) != hipSuccess)
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear device memory for inner product on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				solverStatusDMR = hipsolverDnDestroyGesvdjInfo(geSVjPrmsDMR);

				if(solverStatusDMR != HIPSOLVER_STATUS_SUCCESS)
				{
					fprintf(stderr, "freeDMR CUBLAS Error: Failed to clear SV parameter information on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				solverStatusDMR = hipsolverDnDestroy(slvHandleSVDMR);

				if(solverStatusDMR != HIPSOLVER_STATUS_SUCCESS)
				{
					fprintf(stderr, "freeDMR CUBLAS Error: Failed to clear SV solver handle on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipStreamDestroy(slvStreamSVDMR) != hipSuccess)
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear SV solver stream on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree(cpyHessArrDMR) != hipSuccess)
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear QR work area on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree(tCUSLVDMR) != hipSuccess)
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear second QR work area on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;	
				}

				if(hipFree(wCUSLVDMR) != hipSuccess)
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear third QR work area on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;	
				}

				solverStatusDMR = hipsolverDnDestroy(slvHandleQRDMR);

				if(solverStatusDMR != HIPSOLVER_STATUS_SUCCESS)
				{
					fprintf(stderr, "freeDMR CUSOLVER Error: Failed to clear QR decomposition parameter information on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipStreamDestroy(slvStreamQRDMR) != hipSuccess)
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to clear QR decomposition stream on head solver.\n");
					fullResetDMR(numDevsDMR);
					return;
				}
			}

			if(hipFree((void*) itrBasisDMR[devItr]) != hipSuccess) 
			{
				fprintf(stderr, "freeDMR CUDA Error: Failed to free memory for Krylov basis on device %d.\n", devItr);
				fullResetDMR(numDevsDMR);
				return;
			}

			if(localDefDimDMR > 0)
			{
				if(hipFree((void*) defBasisDMR[devItr]) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to free memory for source side singular vectors on device %d.\n", devItr);
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree((void*) defWorkDMR[devItr]) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to free work area for singular vectors on device %d.\n", devItr);
					fullResetDMR(numDevsDMR);
					return;
				}

				if(hipFree((void*) defCoeffsDMR[devItr]) != hipSuccess) 
				{
					fprintf(stderr, "freeDMR CUDA Error: Failed to free memory for deflation basis coefficients on device %d.\n", devItr);
					fullResetDMR(numDevsDMR);
					return;
				}
			}

			if(hipFree((void*) itrVecDMR[devItr]) != hipSuccess) 
			{
				fprintf(stderr, "freeDMR CUDA Error: Failed to free memory for temporary solutions on device %d.\n", devItr);
				fullResetDMR(numDevsDMR);
				return;
			}

			if(hipFree((void*) kryCoeffsDMR[devItr]) != hipSuccess) 
			{
				fprintf(stderr, "freeDMR CUDA Error: Failed to free memory for Krylov basis coefficients on device %d.\n", devItr);
				fullResetDMR(numDevsDMR);
				return;
			}


			if(hipblasDestroy(blasHandleDMR[devItr]) != HIPBLAS_STATUS_SUCCESS) 
			{
				fprintf(stderr, "freeDMR CUBLAS Error: Failed to free cuBLAS handle on device %d.\n", devItr);
				fullResetDMR(numDevsDMR);
				return;
			}

			if(hipStreamDestroy(blasStreamDMR[devItr]) != hipSuccess)
			{
				fprintf(stderr, "freeDMR CUDA Error: Failed to free cuBLAS stream on device %d.\n", devItr);
				fullResetDMR(numDevsDMR);
				return;
			}
		}
		// Free CPU memory
		free(itrVecDMR);
		free(kryCoeffsDMR);
		free(defCoeffsDMR);
		free(itrBasisDMR);
		free(defBasisDMR);
		free(defWorkDMR);
		free(blasHandleDMR);
		free(blasStreamDMR);
		free(devListDMR);
		return;
	}
}
// Sums all itrVec work vectors, and stores the result on the itrVec of the primary solver device. 
// devItrUB is the device iterator upper bound. 
extern "C"{__host__
	int sumItrVecsDMR(int devItrUB)
	{

		for(int devItr = 1; devItr < devItrUB; devItr++)
		{
			if(hipMemcpyPeerAsync(&(sumWorkVecDMR[(devItr - 1) * numElementsDMR]), devListDMR[0], itrVecDMR[devItr], devListDMR[devItr], vecSizeDMR, blasStreamDMR[devItr]) != hipSuccess)
			{
				fprintf (stderr, "sumItrVecsDMR CUDA Error: Failed to copy coefficient information to device %d.\n", devItr);
				fullResetDMR(numDevsDMR);
				return 1;
			}
		}

		for(int devItr = 0; devItr < devItrUB; devItr++)
		{
			if(hipStreamSynchronize(blasStreamDMR[devItr]) != hipSuccess)
			{
				fprintf(stderr, "sumItrVecsDMR CUDA Error: Failed to synchronize BLAS stream %d.\n", devItr);
				fullResetDMR(numDevsDMR);
				return 1;
			}
		}

		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "sumItrVecsDMR CUDA Error: Failed to set device to head solver.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		vecAddArrDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(devItrUB - 1, numElementsDMR, itrVecDMR[0], sumWorkVecDMR);

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "sumItrVecsDMR CUDA Error: Failed to synchronize BLAS stream 0.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		return 0;	
	}
}
// Loads vector components from each basis segment into the per device itrVec. 
// Vectors are generated over the entire basis, ie. both Krylov and deflation vectors
// are included. Mode -1 uses the source space basis, all other numbers use the target side basis.
extern "C"{__host__
	int genItrVecsDMR(int mode, int defDim, hipComplex *coeffs)
	{
		int localDimDef = (defDim == 0) ? 0 : localDefDimDMR;
		// Copy coefficients and sum results into iteration vectors.
		for(int devItr = 0; devItr < numDevsDMR; devItr++)
		{
			if(hipSetDevice(devListDMR[devItr]) != hipSuccess)
			{
				fprintf(stderr, "genItrVecsDMR CUDA Error: Failed to switch to device %d.\n", devItr);
				fullResetDMR(numDevsDMR);
				return 1;
			}

			zeroArrayDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[devItr]>>>(0, numElementsDMR, itrVecDMR[devItr]);
			// Krylov vector components.
			hipMemcpyPeerAsync(kryCoeffsDMR[devItr], devListDMR[devItr], &(coeffs[defDim + devItr * (localBasDimDMR - localDimDef)]), devListDMR[0], sizeof(hipComplex) * (localBasDimDMR - localDimDef), blasStreamDMR[devItr]);
			// Sum basis coefficients into device itrVecDMR.
			addBasisVecsDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[devItr]>>>(localBasDimDMR - localDimDef, numElementsDMR, kryCoeffsDMR[devItr], &(itrBasisDMR[devItr][localDimDef * numElementsDMR]), itrVecDMR[devItr]);
			// Deflation vector components.
			if(localDimDef > 0)
			{
				hipMemcpyPeerAsync(defCoeffsDMR[devItr], devListDMR[devItr], &(coeffs[devItr * localDimDef]), devListDMR[0], sizeof(hipComplex) * localDimDef, blasStreamDMR[devItr]);
				// Sum for source side vectors.
				if(mode == -1)
				{
					addBasisVecsDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[devItr]>>>(localDimDef, numElementsDMR, defCoeffsDMR[devItr], defBasisDMR[devItr], itrVecDMR[devItr]);
				}
				// Sum for target side vectors.
				else
				{
					addBasisVecsDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[devItr]>>>(localDimDef, numElementsDMR, defCoeffsDMR[devItr], itrBasisDMR[devItr], itrVecDMR[devItr]);
				}
			}
		}
		// Synchronize streams before moving to summing step
		for(int devItr = 0; devItr < numDevsDMR; devItr++)
		{
			if(hipStreamSynchronize(blasStreamDMR[devItr]) != hipSuccess)
			{
				fprintf(stderr, "genItrVecsDMR CUDA Error: Failed to synchronize BLAS stream %d after vector generation.\n", devItr);
				fullResetDMR(numDevsDMR);
				return 1;
			}		
		}
		// Sum all iteration vector together, storing the result in itrVecDMR[0]
		if(numDevsDMR > 1)
		{	
			if(sumItrVecsDMR(numDevsDMR) != 0)
			{
				return 1;
			}
		}

		return 0;
	}
}
// Generate source side deflation vectors. Mode zero corresponds to a first run of the Arnoldi 
// algorithm where no deflation vectors are used. 
extern "C"{__host__
	int genDeflatDMR(int mode)
	{
		int devLoc, memLoc, defDim, localDefDim;

		if(mode == 0)
		{
			localDefDim = 0;
			defDim = 0;
		}
		else
		{
			localDefDim = localDefDimDMR;
			defDim = deflateDimDMR; 
		}
		// Perform svd decompositions. 
		if(svDecompHessDMR() != 0)
		{
			return 1;
		}
		
		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "genDeflatDMR CUDA Error: Failed to set device.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		zeroArrayDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(0, basisDimDMR + 1, hessTrgDMR);

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "genDeflatDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}
		// Source side
		for(int vecItr = 0; vecItr < deflateDimDMR; vecItr++)
		{
			devLoc = vecItr / localDefDimDMR;
			memLoc = vecItr % localDefDimDMR;

			if(hipSetDevice(devListDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "genDeflatDMR CUDA Error: Failed to set device.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			if(hipMemcpyPeerAsync(hessWrkDMR, devListDMR[0], &(vHessArrDMR[(vecItr + basisDimDMR - deflateDimDMR) * basisDimDMR]), devListDMR[0], sizeof(hipComplex) * basisDimDMR, blasStreamDMR[0]) != hipSuccess)
			{
				fprintf (stderr, "genDeflatDMR CUDA Error: Failed to copy coefficient information to work vector.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}
			// Reorder coefficients
			reorderHessCoeffs<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(1, numDevsDMR, defDim, basisDimDMR + 1, localBasDimDMR - localDefDim, hessWrkDMR, hessTrgDMR);

			if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "genDeflatDMR CUDA Error: Failed to reorder coefficients.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}
			// Compute vector
			if(genItrVecsDMR(-1, defDim, hessTrgDMR) != 0)
			{
				return 1;
			}

			if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "genDeflatDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}
			// Normalize
			if(innerProdDMR(numElementsDMR, itrVecDMR[0], itrVecDMR[0], innProdDevDMR, devListDMR[0]) != 0)
			{
				return 1;
			}
			
			normalizeVecDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(-1, numElementsDMR, itrVecDMR[0], innProdDevDMR, 0.0);
			
			if(hipMemcpyPeerAsync(&(defWorkDMR[devLoc][memLoc * numElementsDMR]), devListDMR[devLoc], itrVecDMR[0], devListDMR[0], vecSizeDMR, blasStreamDMR[0]) != hipSuccess)
			{
				fprintf (stderr, "genDeflatDMR CUDA Error: Failed to copy coefficient information to device %d.\n", devLoc);
				fullResetDMR(numDevsDMR);
				return 1;
			}
			
			if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "genDeflatDMR CUDA Error: Failed to synchronize solver stream.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}
		}
		// Update all source vectors: move vectors from work area into storage area.  
		for(int devItr = 0; devItr < numDevsDMR; devItr++)
		{
			if(hipMemcpyPeerAsync(defBasisDMR[devItr], devListDMR[devItr], defWorkDMR[devItr], devListDMR[devItr], vecSizeDMR * localDefDimDMR, blasStreamDMR[devItr]) != hipSuccess)
			{
				fprintf (stderr, "genDeflatDMR CUDA Error: Failed to copy singular vectors from work area to storage area on device %d.\n", devItr);
				fullResetDMR(numDevsDMR);
				return 1;
			}
		}
		// Synchronize devices
		for(int devItr = 0; devItr < numDevsDMR; devItr++)
		{
			if(hipStreamSynchronize(blasStreamDMR[devItr]) != hipSuccess)
			{
				fprintf(stderr, "genDeflatDMR CUDA Error: Failed to synchronize BLAS stream %d.\n", devItr);
				fullResetDMR(numDevsDMR);
				return 1;
			}
		}

		return 0;
	}
}
// Operates in place, deflating the vector held in itrVecDMR[0]. saveMode != 0 saves 
// projections to the Hessenberg matrix based on the Krylov iteration kryNum but does not
// renormalize the vector. 
// saveMode == 0,  orthogonalizes the vector with respect to the deflation basis, but does not
// save the projection components.
extern "C"{__host__
	int deflatorDMR(int saveMode, int kryNum) 
	{
		int hessenPos;

		for(int devItr = 1; devItr < numDevsDMR; devItr++)
		{
			if(hipMemcpyPeerAsync(itrVecDMR[devItr], devListDMR[devItr], itrVecDMR[0], devListDMR[0], vecSizeDMR, blasStreamDMR[devItr]) != hipSuccess)
			{
				fprintf (stderr, "deflatorDMR CUDA Error: Failed to copy inflated vector to device %d.\n", devItr);
				fullResetDMR(numDevsDMR);
				return 1;
			}	
		}

		for(int devItr = 0; devItr < numDevsDMR; devItr++)
		{
			if(basisProjDMR(itrBasisDMR[devItr], itrVecDMR[devItr], defCoeffsDMR[devItr], localDefDimDMR, numElementsDMR, devItr) != 0)
			{
				return 1;
			}
			// Position in Hessenberg array
			if(saveMode != 0)
			{
				hessenPos = (deflateDimDMR + kryNum - 1) * (basisDimDMR + 1) + devItr * localDefDimDMR;
				// Save results to Hessenberg matrix
				hipMemcpyPeerAsync(&(hessArrDMR[hessenPos]), devListDMR[0], defCoeffsDMR[devItr], devListDMR[devItr], sizeof(hipComplex) * localDefDimDMR, blasStreamDMR[devItr]);
			}
			// Remove deflation components
			if(hipSetDevice(devListDMR[devItr]) != hipSuccess)
			{
				fprintf(stderr, "deflatorDMR CUDA Error: Failed to set device.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			vecScaleDiffDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[devItr]>>>(localDefDimDMR, numElementsDMR, defCoeffsDMR[devItr], itrBasisDMR[devItr], 1.0 / numDevsDMR, itrVecDMR[devItr]);
		}

		for(int devItr = 0; devItr < numDevsDMR; devItr++)
		{
			if(hipStreamSynchronize(blasStreamDMR[devItr]) != hipSuccess)
			{
				fprintf(stderr, "deflatorDMR CUDA Error: Failed to synchronize BLAS stream %d.\n", devItr);
				fullResetDMR(numDevsDMR);
				return 1;
			}
		}

		if(numDevsDMR > 1)
		{
			if(sumItrVecsDMR(numDevsDMR) != 0)
			{
				return 1;
			}
		}

		if(saveMode == 0)
		{
			if(innerProdDMR(numElementsDMR, itrVecDMR[0], itrVecDMR[0], innProdDevDMR, devListDMR[0]) != 0)
			{
				return 1;
			}

			normalizeVecDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(-1, numElementsDMR, itrVecDMR[0], innProdDevDMR, 0.0);

			if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "deflatorDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

		}

		return 0;	
	}
}
// Arnoldi algorithm, building the Krylov search space. If a defect vector is generated,
// the program zeros all remaining basis locations and exits.
extern "C"{__host__
	int arnoldiDMR(int dimDeflateSpace, hipComplex *extSrcVec, hipStream_t extStream, int extDevNum, float defectTol)
	{	
		int prodsPerDev, offsetDevs;
		int filledLoc = 0;
		int loopBound = 0;
		int hessenPos;
		int localDefDim;
		hipComplex innerProd;
		float norm;
		int devNumS;
		int memNumS;
		int devNumT;
		int memNumT;

		localDefDim = (dimDeflateSpace > 0) ? localDefDimDMR : 0;

		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "arnoldiDMR CUDA Error: Failed to set device.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}
		// Zero Hessenberg matrix 
		zeroArrayDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(0, (basisDimDMR + 1) * basisDimDMR, hessArrDMR);

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}
		// Compute deflation space. 
		for(int deflatItr = 0; deflatItr < dimDeflateSpace; deflatItr++)
		{
			devNumS = deflatItr / localDefDimDMR;
			memNumS = deflatItr % localDefDimDMR;
			devNumT = deflatItr % numDevsDMR;
			memNumT = deflatItr / numDevsDMR;

			loopBound = (memNumT > 0) ? numDevsDMR : devNumT;

			if(hipMemcpyPeerAsync(extSrcVec, extDevNum, &(defBasisDMR[devNumS][memNumS * numElementsDMR]), devListDMR[devNumS], vecSizeDMR, extStream) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to copy source deflation vector %d, to linear operator device.\n", devNumS * localDefDimDMR + memNumS + 1);
				fullResetDMR(numDevsDMR);
				return 1;
			}

			if(hipSetDevice(extDevNum) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to set device to global linear operator.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			(*linOptDMR)();
			
			if(hipMemcpyPeerAsync(itrVecDMR[0], devListDMR[0], extSrcVec, extDevNum, vecSizeDMR, extStream) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to copy global vector to head solver.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			if(hipStreamSynchronize(extStream) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize global stream.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			if(hipMemcpyPeerAsync(genWorkVecDMR, devListDMR[0], extSrcVec, extDevNum, vecSizeDMR, blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to external vector to general workspace.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}
			
			for(int devItr = 1; devItr < loopBound; devItr++)
			{
				if(hipMemcpyPeerAsync(itrVecDMR[devItr], devListDMR[devItr], itrVecDMR[0], devListDMR[0], vecSizeDMR, blasStreamDMR[devItr]) != hipSuccess)
				{
					fprintf(stderr, "arnoldiDMR CUDA Error: Failed to copy iteration vector to device %d.\n", devListDMR[devItr]);
					fullResetDMR(numDevsDMR);
					return 1;
				}	
			}
			// Compute Hessenberg matrix coefficients.
			for(int devItr = 0; devItr < loopBound; devItr++)
			{
				filledLoc = (devItr < devNumT) ? (memNumT + 1) : memNumT;

				if(hipSetDevice(devListDMR[devItr]) != hipSuccess)
				{
					fprintf(stderr, "arnoldiDMR CUDA Error: Failed to set device to %d.\n", devItr);
					fullResetDMR(numDevsDMR);
					return 1;
				}

				zeroArrayDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[devItr]>>>(0, localDefDimDMR, defCoeffsDMR[devItr]);

				if(basisProjDMR(defWorkDMR[devItr], itrVecDMR[devItr], defCoeffsDMR[devItr], filledLoc, numElementsDMR, devItr) != 0)
				{
					return 1;
				}
			}
			// Copy deflation coefficients into Hessenberg matrix.
			if(hipSetDevice(devListDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to set device to head solver.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			zeroArrayDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(0, basisDimDMR + 1, hessWrkDMR);

			for(int devItr = 0; devItr < loopBound; devItr++)
			{
				if(hipStreamSynchronize(blasStreamDMR[devItr]) != hipSuccess)
				{
					fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize BLAS stream %d.\n", devItr);
					fullResetDMR(numDevsDMR);
					return 1;
				}
			}

			for(int devItr = 0; devItr < loopBound; devItr++)
			{
				filledLoc = (devItr < devNumT) ? (memNumT + 1) : memNumT;

				hipMemcpyPeerAsync(&(hessWrkDMR[devItr * localDefDimDMR]), devListDMR[0], defCoeffsDMR[devItr], devListDMR[devItr], sizeof(hipComplex) * localDefDimDMR, blasStreamDMR[devItr]);

				if(hipSetDevice(devListDMR[devItr]) != hipSuccess)
				{
					fprintf(stderr, "arnoldiDMR CUDA Error: Failed to set device.\n");
					fullResetDMR(numDevsDMR);
					return 1;
				}
				// Scale iteration vectors by number of devices and remove Krylov components
				vecScaleDiffDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[devItr]>>>(filledLoc, numElementsDMR, defCoeffsDMR[devItr], defWorkDMR[devItr], 1.0 / loopBound, itrVecDMR[devItr]);
			}

			for(int devItr = 0; devItr < loopBound; devItr++)
			{
				if(hipStreamSynchronize(blasStreamDMR[devItr]) != hipSuccess)
				{
					fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize BLAS stream %d.\n", devItr);
					fullResetDMR(numDevsDMR);
					return 1;
				}
			}
			
			if(loopBound > 1)
			{
				if(sumItrVecsDMR(loopBound) != 0)
				{
					return 1;
				}
			}
			// Load results into Hessenberg
			hessenPos = deflatItr * (basisDimDMR + 1);

			reorderHessCoeffs<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(-1, numDevsDMR, 0, dimDeflateSpace, localDefDim, hessWrkDMR, &(hessArrDMR[hessenPos]));

			if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}
			// Normalize and Compute final Hessenberg component of new basis vector
			if(innerProdDMR(numElementsDMR, itrVecDMR[0], itrVecDMR[0], innProdDevDMR, devListDMR[0]) != 0)
			{
				return 1;
			}

			if(hipSetDevice(devListDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to set device.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			normalizeVecDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(-1, numElementsDMR, itrVecDMR[0], innProdDevDMR, 0.0);

			if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}
			// Add vector to target basis 
			if(hipMemcpyPeerAsync(&(defWorkDMR[devNumT][memNumT * numElementsDMR]), devListDMR[devNumT], itrVecDMR[0], devListDMR[0], vecSizeDMR, blasStreamDMR[devNumT]) != hipSuccess)
			{
				fprintf (stderr, "arnoldiDMR CUDA Error: Failed to copy new vector into basis.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}			

			if(hipMemcpyPeerAsync(&(itrBasisDMR[devNumS][memNumS * numElementsDMR]), devListDMR[devNumS], itrVecDMR[0], devListDMR[0], vecSizeDMR, blasStreamDMR[devNumS]) != hipSuccess)
			{
				fprintf (stderr, "arnoldiDMR CUDA Error: Failed to copy new vector into basis.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}
			// Compute final Hessenberg entry
			hessenPos = (deflatItr) * (basisDimDMR + 1) + deflatItr;
			// Inner product of original image with new vector
			if(innerProdDMR(numElementsDMR, itrVecDMR[0], genWorkVecDMR, &(hessArrDMR[hessenPos]), devListDMR[0]) != 0)
			{
				return 1;
			}

			if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			if(hipStreamSynchronize(blasStreamDMR[devNumS]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize BLAS stream %d.\n", devNumS);
				fullResetDMR(numDevsDMR);
				return 1;
			}

			if(hipStreamSynchronize(blasStreamDMR[devNumT]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize BLAS stream %d.\n", devNumT);
				fullResetDMR(numDevsDMR);
				return 1;
			}
		}
		// Orthogonalize starting Krylov vector with respect to deflation basis.
		if(dimDeflateSpace > 0)
		{
			if(hipMemcpyPeerAsync(itrVecDMR[0], devListDMR[0], &(itrBasisDMR[0][localDefDim * numElementsDMR]), devListDMR[0], vecSizeDMR, blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to copy starting Krylov vector.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}
		
			if(deflatorDMR(0, 0) != 0)
			{
				return 1;
			}

			if(hipMemcpyPeerAsync(&(itrBasisDMR[0][localDefDim * numElementsDMR]), devListDMR[0], itrVecDMR[0], devListDMR[0], vecSizeDMR, blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to copy starting Krylov vector.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}
		}
		// Switch to Krylov basis computation
		if(hipMemcpyPeerAsync(extSrcVec, extDevNum, &(itrBasisDMR[0][localDefDim * numElementsDMR]), devListDMR[0], vecSizeDMR, extStream) != hipSuccess)
		{
			fprintf(stderr, "arnoldiDMR CUDA Error: Failed to copy initial Krylov vector.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		for(int krylovItr = 1; krylovItr < (basisDimDMR - dimDeflateSpace + 1); krylovItr++)
		{
			if(hipSetDevice(extDevNum) != hipSuccess)
			{
				fprintf(stderr, "deflatorDMR CUDA Error: Failed to set device to head linear operator.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			(*linOptDMR)();
			
			if(hipMemcpyPeerAsync(itrVecDMR[0], devListDMR[0], extSrcVec, extDevNum, vecSizeDMR, extStream) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to copy global vector to solver.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			if(hipStreamSynchronize(extStream) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize global stream.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			if(hipSetDevice(devListDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "deflatorDMR CUDA Error: Failed to set device to head solver.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			if(hipMemcpyPeerAsync(genWorkVecDMR, devListDMR[0], itrVecDMR[0], devListDMR[0], vecSizeDMR, blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to copy iteration vector to workspace.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to head BLAS stream.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}		
			// Deflate iterated vector, fills upper right hand block of Hessenberg matrix.
			if(dimDeflateSpace > 0)
			{	
				if(deflatorDMR(1, krylovItr) != 0)
				{
					return 1;
				}
			}			

			prodsPerDev = krylovItr / numDevsDMR;
			offsetDevs = krylovItr % numDevsDMR;
			loopBound = (prodsPerDev > 0) ? numDevsDMR : offsetDevs;

			for(int devItr = 1; devItr < loopBound; devItr++)
			{
				
				if(hipMemcpyPeerAsync(itrVecDMR[devItr], devListDMR[devItr], itrVecDMR[0], devListDMR[0], vecSizeDMR, blasStreamDMR[devItr]) != hipSuccess)
				{
					fprintf(stderr, "arnoldiDMR CUDA Error: Failed to copy iteration vector to device %d.\n", devListDMR[devItr]);
					fullResetDMR(numDevsDMR);
					return 1;
				}	
			}

			for(int devItr = 1; devItr < loopBound; devItr++)
			{
				if(hipStreamSynchronize(blasStreamDMR[devItr]) != hipSuccess)
				{
					fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize BLAS stream %d.\n", devItr);
					fullResetDMR(numDevsDMR);
					return 1;
				}
			}
			// Compute Hessenberg matrix coefficients with Arnoldi generated vectors.
			// Deflation vectors handled separately by deflatorDMR.
			for(int devItr = 0; devItr < loopBound; devItr++)
			{
				filledLoc = (devItr < offsetDevs) ? (prodsPerDev + 1) :	prodsPerDev;
				
				if(hipSetDevice(devListDMR[devItr]) != hipSuccess)
				{
					fprintf(stderr, "arnoldiDMR CUDA Error: Failed to set device to %d.\n", devItr);
					fullResetDMR(numDevsDMR);
					return 1;
				}

				zeroArrayDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[devItr]>>>(0, localBasDimDMR, kryCoeffsDMR[devItr]);

				if(basisProjDMR(&(itrBasisDMR[devItr][localDefDim * numElementsDMR]), itrVecDMR[devItr], kryCoeffsDMR[devItr], filledLoc, numElementsDMR, devItr) != 0)
				{
					return 1;
				}
			}
			// Copy Krylov coefficients into Hessenberg matrix.
			if(hipSetDevice(devListDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to set device to head solver.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			zeroArrayDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(0, basisDimDMR + 1, hessWrkDMR);

			if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize BLAS stream 0.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			for(int devItr = 0; devItr < loopBound; devItr++)
			{
				filledLoc = (devItr < offsetDevs) ? (prodsPerDev + 1) :	prodsPerDev;

				if(hipSetDevice(devListDMR[devItr]) != hipSuccess)
				{
					fprintf(stderr, "arnoldiDMR CUDA Error: Failed to set device.\n");
					fullResetDMR(numDevsDMR);
					return 1;
				}
				
				hipMemcpyPeerAsync(&(hessWrkDMR[devItr * (localBasDimDMR - localDefDim)]), devListDMR[0], kryCoeffsDMR[devItr], devListDMR[devItr], sizeof(hipComplex) * (localBasDimDMR - localDefDim), blasStreamDMR[devItr]);
				// Scale iteration vectors by number of devices and remove Krylov components
				vecScaleDiffDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[devItr]>>>(filledLoc, numElementsDMR, kryCoeffsDMR[devItr], &(itrBasisDMR[devItr][localDefDim * numElementsDMR]), 1.0 / loopBound, itrVecDMR[devItr]);
			}

			for(int devItr = 0; devItr < loopBound; devItr++)
			{
				if(hipStreamSynchronize(blasStreamDMR[devItr]) != hipSuccess)
				{
					fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize BLAS stream %d.\n", devItr);
					fullResetDMR(numDevsDMR);
					return 1;
				}
			}
			
			if(loopBound > 1)
			{
				if(sumItrVecsDMR(loopBound) != 0)
				{
					return 1;
				}
			}
			// Load results into Hessenberg
			if(hipSetDevice(devListDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to set device.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			hessenPos = (dimDeflateSpace + krylovItr - 1) * (basisDimDMR + 1) + dimDeflateSpace;

			reorderHessCoeffs<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(-1, numDevsDMR, 0, basisDimDMR - dimDeflateSpace, localBasDimDMR - localDefDim, hessWrkDMR, &(hessArrDMR[hessenPos]));

			if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}
			// Normalize iteration vector and compute final Hessenberg component of new basis vector
			if(innerProdDMR(numElementsDMR, itrVecDMR[0], itrVecDMR[0], innProdDevDMR, devListDMR[0]) != 0)
			{
				return 1;
			}

			if(hipSetDevice(devListDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to set device.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			if(hipMemcpy(&innerProd, innProdDevDMR, sizeof(hipComplex), hipMemcpyDeviceToHost) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to copy inner product to host.\n");
				return 1;	
			}

			norm = sqrtf(innerProd.x);

			if(norm < defectTol)
			{
				for(int devItr = 0; devItr < numDevsDMR; devItr++)
				{
					filledLoc = (devItr < offsetDevs) ? (prodsPerDev + 1) :	prodsPerDev;
					
					if(hipSetDevice(devListDMR[devItr]) != hipSuccess)
					{
						fprintf(stderr, "arnoldiDMR CUDA Error: Failed to set device.\n");
						fullResetDMR(numDevsDMR);
						return 1;
					}

					zeroArrayDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[devItr]>>>((localDefDim + filledLoc) * numElementsDMR, localBasDimDMR * numElementsDMR, itrBasisDMR[devItr]);

					if(devItr == 0)
					{
						zeroArrayDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[devItr]>>>(0, numElementsDMR, itrVecDMR[devItr]);
						// Make Hessenberg full rank for QR decomposition.
						eyeArrayDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[devItr]>>>(krylovItr * (basisDimDMR + 1), basisDimDMR + 1, basisDimDMR, hessArrDMR);
					} 
				}

				for(int devItr = 0; devItr < numDevsDMR; devItr++)
				{
					if(hipStreamSynchronize(blasStreamDMR[devItr]) != hipSuccess)
					{
						fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize BLAS stream %d after zeroing solution basis.\n", devItr);
						fullResetDMR(numDevsDMR);
						return 1;
					}
				}

				return 0;
			}

			normalizeVecDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(-1, numElementsDMR, itrVecDMR[0], innProdDevDMR, 0.0);
			// Add vector to Krylov basis 
			if(hipMemcpyPeerAsync(&(itrBasisDMR[offsetDevs][(localDefDim + prodsPerDev) * numElementsDMR]), devListDMR[offsetDevs], itrVecDMR[0], devListDMR[0], vecSizeDMR, blasStreamDMR[0]) != hipSuccess)
			{
				fprintf (stderr, "arnoldiDMR CUDA Error: Failed to copy new vector into basis.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}
			// Compute final Hessenberg entry
			hessenPos = (dimDeflateSpace + krylovItr - 1) * (basisDimDMR + 1) + dimDeflateSpace + krylovItr;
			// Inner product of original image with new vector
			if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			if(innerProdDMR(numElementsDMR, itrVecDMR[0], genWorkVecDMR, &(hessArrDMR[hessenPos]), devListDMR[0]) != 0)
			{
				return 1;
			}

			if(hipMemcpyPeerAsync(extSrcVec, extDevNum, itrVecDMR[0], devListDMR[0], vecSizeDMR, extStream) != hipSuccess)
			{
				fprintf (stderr, "arnoldiDMR CUDA Error: Failed to copy to linear computer into basis.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}

			if(hipStreamSynchronize(extStream) != hipSuccess)
			{
				fprintf(stderr, "arnoldiDMR CUDA Error: Failed to synchronize external stream.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}
		}

		return 0;
	}
}

extern "C"{__host__
	int updateTrgCoeffsDMR(int defDim)
	{
		int localDefDim;

		localDefDim = (defDim == 0) ? 0 : localDefDimDMR;

		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "updateTrgCoeffsDMR CUDA Error: Failed to set device to head solver.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		zeroArrayDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(0, basisDimDMR + 1, hessWrkDMR);

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "updateTrgCoeffsDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		for(int devItr = 0; devItr < numDevsDMR; devItr++)
		{
			// Copy residual vector to all devices
			hipMemcpyPeerAsync(itrVecDMR[devItr], devListDMR[devItr], resVecDMR, devListDMR[0], vecSizeDMR, blasStreamDMR[devItr]);
			// Calculate projection coefficients
			if(defDim != 0)
			{
				if(basisProjDMR(itrBasisDMR[devItr], itrVecDMR[devItr], defCoeffsDMR[devItr], localDefDim, numElementsDMR, devItr) != 0)
				{
					return 1;
				}

				hipMemcpyPeerAsync(&(hessWrkDMR[devItr * localDefDim]), devListDMR[0], defCoeffsDMR[devItr], devListDMR[devItr], sizeof(hipComplex) * localDefDim, blasStreamDMR[devItr]);			
			}

			if(basisProjDMR(&(itrBasisDMR[devItr][localDefDim * numElementsDMR]), itrVecDMR[devItr], kryCoeffsDMR[devItr], localBasDimDMR - localDefDim, numElementsDMR, devItr) != 0)
			{
				return 1;
			}

			hipMemcpyPeerAsync(&(hessWrkDMR[defDim + devItr * (localBasDimDMR - localDefDim)]), devListDMR[0], kryCoeffsDMR[devItr], devListDMR[devItr], sizeof(hipComplex) * (localBasDimDMR - localDefDim), blasStreamDMR[devItr]);
		}

		for(int devItr = 0; devItr < numDevsDMR; devItr++)
		{
			if(hipStreamSynchronize(blasStreamDMR[devItr]) != hipSuccess)
			{
				fprintf(stderr, "updateTrgCoeffsDMR CUDA Error: Failed to synchronize BLAS stream %d.\n", devItr);
				fullResetDMR(numDevsDMR);
				return 1;
			}
		}
		// Load new projCoeffs
		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "updateTrgCoeffsDMR CUDA Error: Failed to set device.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		reorderHessCoeffs<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(-1, numDevsDMR, defDim, basisDimDMR + 1, localBasDimDMR - localDefDim, hessWrkDMR, hessPrjDMR);
		
		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "updateTrgCoeffsDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}
		
		return 0;
	}
}
// Updates the residual, approximate solution, and global vectors using the 
// results of the current solve. Outputs the current norm of the residual. 
extern "C"{__host__
	float updateAppResDMR(hipComplex *extSrcVec, hipStream_t extStream, int extDevNum, int defDim, float solTol, float prevNorm)
	{
		hipComplex resProdHost;
		float resNorm = 0;
		int localDimDef;
		
		localDimDef = (defDim == 0) ? 0 : localDefDimDMR;
		// Calculate target coefficients. 
		if(updateTrgCoeffsDMR(defDim) != 0)
		{
			return prevNorm;
		}
		// Calculate coefficients of the residual.
		if(qrHessArrDMR() != 0)
		{
			return prevNorm;
		}

		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "updateAppResDMR CUDA Error: Failed to set device.\n");
			fullResetDMR(numDevsDMR);
			return prevNorm;
		}

		if(hipMemcpy(hessWrkDMR, hessPrjDMR, sizeof(hipComplex) * (basisDimDMR + 1), hipMemcpyDeviceToDevice) != hipSuccess)
		{
			fprintf(stderr, "updateAppResDMR CUDA Error: Failed to copy hessPrjDMR into coefficient vector.\n");
			fullResetDMR(numDevsDMR);
			return prevNorm;
		}

		if(qrSolveDMR(basisDimDMR + 1, basisDimDMR, hessWrkDMR, hessSrcDMR, qHessArrDMR, rHessArrDMR) != 0)
		{
			return prevNorm;
		}
		// Hessenberg matrix acts on hessSrcDMR vector and places result in hessTrgDMR.
		if(hessenbergMultDMR() != 0)
		{
			return prevNorm;
		}
		// Reorder coefficients
		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "updateAppResDMR CUDA Error: Failed to set device.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		reorderHessCoeffs<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(1, numDevsDMR, defDim, basisDimDMR + 1, localBasDimDMR - localDimDef, hessWrkDMR, hessTrgDMR);

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "updateAppResDMR CUDA Error: Failed to synchronize BLAS stream %d after coefficient reordering.\n", 0);
			fullResetDMR(numDevsDMR);
			return 1;
		}
		// Update residual
		if(genItrVecsDMR(1, defDim, hessTrgDMR) != 0)
		{
			return prevNorm;
		}
		// Add update to previous residual
		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "updateAppResDMR basisProjDMR CUDA Error: Failed to set device.\n");
			fullResetDMR(numDevsDMR);
			return prevNorm;
		}

		vecAddDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>> (-1, numElementsDMR, resVecDMR, itrVecDMR[0]);

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "updateAppResDMR CUDA Error: Failed to synchronize main BLAS stream.\n");
			fullResetDMR(numDevsDMR);
			return prevNorm;
		}
		// Compute norm change
		if(innerProdDMR(numElementsDMR, resVecDMR, resVecDMR, innProdDevDMR, devListDMR[0]) != 0)
		{
			return prevNorm;
		}

		if(hipMemcpy(&resProdHost, innProdDevDMR, sizeof(hipComplex), hipMemcpyDeviceToHost) != hipSuccess)
		{
			fprintf(stderr, "updateAppResDMR CUDA Error: Failed to copy residual to host.\n");
			fullResetDMR(numDevsDMR);
			return prevNorm;
		}

		resNorm = sqrtf(resProdHost.x);
		// Set Krylov loop flag if change is smaller than solution tolerance. This switch initial 
		// vector for constructing the Krylov basis.
		if(resNorm > 0.995)
		{
			kryLoopFlagDMR = 1;
			kryFlagCountDMR++;
		}
		else
		{
			kryLoopFlagDMR = 0;
		}
		// Renormalize residual vector
		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "updateAppResDMR CUDA Error: Failed to set device.\n");
			fullResetDMR(numDevsDMR);
			return prevNorm;
		}

		normalizeVecDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(-1, numElementsDMR, resVecDMR, innProdDevDMR, 0.0);

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "updateAppResDMR CUDA Error: Failed to synchronize main BLAS stream.\n");
			fullResetDMR(numDevsDMR);
			return prevNorm;
		}		
		// Update approximate solution
		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "updateAppResDMR basisProjDMR CUDA Error: Failed to set device.\n");
			fullResetDMR(numDevsDMR);
			return prevNorm;
		}

		zeroArrayDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(0, basisDimDMR + 1, hessWrkDMR);

		if(hipMemcpyPeerAsync(hessWrkDMR, devListDMR[0], hessSrcDMR, devListDMR[0], sizeof(hipComplex) * basisDimDMR, blasStreamDMR[0]) != hipSuccess)
		{
			fprintf (stderr, "updateAppResDMR CUDA Error: Failed to copy coefficient information to device.\n");
			fullResetDMR(numDevsDMR);
			return prevNorm;
		}
		// Reorder coefficients
		reorderHessCoeffs<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(1, numDevsDMR, defDim, basisDimDMR + 1, localBasDimDMR - localDimDef, hessWrkDMR, hessTrgDMR);

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "updateAppResDMR CUDA Error: Failed to synchronize BLAS stream %d after coefficient reordering.\n", 0);
			fullResetDMR(numDevsDMR);
			return 1;
		}
		// Update approximate solution.
		if(genItrVecsDMR(-1, defDim, hessTrgDMR) != 0)
		{
			return prevNorm;
		}

		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "updateAppResDMR CUDA Error: Failed to set device.\n");
			fullResetDMR(numDevsDMR);
			return prevNorm;
		}
		// Normalize size of update
		normalizeVecDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(2, numElementsDMR, itrVecDMR[0], innProdDevDMR, prevNorm);
		// Add update to previous approximation
		vecAddDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>> (1, numElementsDMR, aSrcVecDMR, itrVecDMR[0]);

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "updateAppResDMR CUDA Error: Failed to synchronize main BLAS stream.\n");
			fullResetDMR(numDevsDMR);
			return prevNorm;
		}

		return resNorm * prevNorm;
	}
}
// Generate source coefficients and creates an initial vector for the Arnoldi method.
// mode == 0 suppose that there is no deflation space present. For all other numbers, a previously
// calculated deflation space is assumed to be present. 
extern "C" {__host__
	int arnoldiSetupDMR(int mode, int matRank, hipComplex *extSrcVec, hipStream_t extStream, int extDevNum)
	{
		int localDefDim;
		// Fetch starting point for new basis
		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "arnoldiSetupDMR CUDA Error: Failed to set device to head inverse solver.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		localDefDim = (mode == 0) ? 0 : localDefDimDMR;
		
		if(kryLoopFlagDMR == 0 || kryFlagCountDMR % (2 * matRank / (basisDimDMR - deflateDimDMR)) == 0)
		{
			if(hipMemcpyPeerAsync(&(itrBasisDMR[0][localDefDim * numElementsDMR]), devListDMR[0], resVecDMR, devListDMR[0], vecSizeDMR, blasStreamDMR[0]) != hipSuccess)
			{
				fprintf (stderr, "arnoldiSetupDMR CUDA Error: Failed to copy current residual into starting position.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}
		}
		else
		{
			kryBasisLoopsDMR = (basisDimDMR - deflateDimDMR) / 2;
			int devLoc = (kryBasisLoopsDMR) % numDevsDMR;
			int memLoc = (kryBasisLoopsDMR) / numDevsDMR;

			if(hipMemcpyPeerAsync(&(itrBasisDMR[0][localDefDim * numElementsDMR]), devListDMR[0], &(itrBasisDMR[devLoc][(localDefDim + memLoc) * numElementsDMR]), devListDMR[devLoc], vecSizeDMR, blasStreamDMR[0]) != hipSuccess)
			{
				fprintf (stderr, "arnoldiSetupDMR CUDA Error: Failed to final vector into starting position.\n");
				fullResetDMR(numDevsDMR);
				return 1;
			}
		}

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess) 
		{
			fprintf(stderr, "arnoldiSetupDMR CUDA Error: Failed to synchronize global linear operator stream.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		return 0;
	}
}
// Find an approximate solution for x for A x = s given a source s, and a linear computer A. 
// A mode value of !0 indicates that deflation spaces have been previously computed, and that 
// these vectors should be used as a starting point in the current computation. The information
// in extImgVec serves as an initial guess for the approximate solution. This vector is rewritten
// during execution. The solver returns the norm of the residual.
extern "C"{__host__
	float solverDMR(int mode, hipComplex *extSrcVec, hipComplex *extImgVec, hipStream_t extStream, int extDevNum, float solTol, int matRank, int *numIts)
	{
		int grdItr = 0;
		int dimDeflat;
		int startDimDef;
		float resNormI, resNorm;
		hipComplex tempProdHost;
		numIts[0] = matRank * matRank;
		// Set local dimensions if previous deflation space exists.
		if(mode == 0)
		{
			startDimDef = 0;
		}
		else
		{
			startDimDef = localDefDimDMR;
		}
		// Reset Krylov flags in case of previous call without reinitialization.
		kryLoopFlagDMR = 0;
		kryBasisLoopsDMR = 0;

		if(hipSetDevice(extDevNum) != hipSuccess)
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to switch to linear operator device.\n");
			fullResetDMR(numDevsDMR);
			return 1.0;
		}

		if(hipDeviceSynchronize() != hipSuccess)
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to synchronize linear operator device.\n");
			fullResetDMR(numDevsDMR);
			return 1.0;
		}
		
		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to set device to head inverse solver.\n");
			fullResetDMR(numDevsDMR);
			return 1.0;
		}

		if(hipMemcpyPeerAsync(resVecDMR, devListDMR[0], extImgVec, extDevNum, vecSizeDMR, blasStreamDMR[0]) != hipSuccess)
		{
			fprintf (stderr, "solverDMR CUDA Error: Failed to copy image into residual.\n");
			fullResetDMR(numDevsDMR);
			return 1.0;
		}

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to synchronize BLAS stream 0.\n");
			fullResetDMR(numDevsDMR);
			return 1.0;
		}
		
		if(innerProdDMR(numElementsDMR, resVecDMR, resVecDMR, innProdDevDMR, devListDMR[0]) != 0)
		{
			fprintf(stderr, "solverDMR Error: innerProdDMR exit.\n");
			return 1.0;
		}
		
		if(hipMemcpy(&tempProdHost, innProdDevDMR, sizeof(hipComplex), hipMemcpyDeviceToHost) != hipSuccess)
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to copy residual to host.\n");
			return 1.0;	
		}
		
		resNormI = sqrtf(tempProdHost.x);
		// Set initial approximation to guess.
		if(hipMemcpyPeerAsync(aSrcVecDMR, devListDMR[0], extSrcVec, extDevNum, vecSizeDMR, blasStreamDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to copy initial guess into approximation.\n");
			fullResetDMR(numDevsDMR);
			return resNormI;
		}

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess) 
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
			fullResetDMR(numDevsDMR);
			return resNormI;
		}
		// Compute initial residual, and save the result as the starting point for the 
		// first application of the Arnoldi algorithm.
		// Apply linear operator to the global vector.
		if(hipSetDevice(extDevNum) != hipSuccess)
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to set device to linear operator.\n");
			fullResetDMR(numDevsDMR);
			return 1;
		}

		(*linOptDMR)();

		if(hipStreamSynchronize(extStream) != hipSuccess)
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to synchronize external stream.\n");
			fullResetDMR(numDevsDMR);
			return 1.0;
		}

		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to set device to head inverse solver.\n");
			fullResetDMR(numDevsDMR);
			return resNormI;
		}

		if(hipMemcpyPeerAsync(itrVecDMR[0], devListDMR[0], extSrcVec, extDevNum, vecSizeDMR, blasStreamDMR[0]) != hipSuccess)
		{
			fprintf (stderr, "solverDMR CUDA Error: Failed to copy initial guess into approximation.\n");
			fullResetDMR(numDevsDMR);
			return resNormI;
		}

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess) 
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
			fullResetDMR(numDevsDMR);
			return resNormI;
		}
		// Find initial residual.
		vecAddDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(-1, numElementsDMR, resVecDMR, itrVecDMR[0]);

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess) 
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to synchronize head BLAS stream.\n");
			fullResetDMR(numDevsDMR);
			return resNormI;
		}
		// Normalize the residual and save the result.
		if(innerProdDMR(numElementsDMR, resVecDMR, resVecDMR, innProdDevDMR, devListDMR[0]) != 0)
		{
			fprintf(stderr, "solverDMR Error: innerProdDMR exit.\n");
			return resNormI;
		}

		if(hipMemcpy(&tempProdHost, innProdDevDMR, sizeof(hipComplex), hipMemcpyDeviceToHost) != hipSuccess)
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to copy residual to host.\n");
			return resNormI;	
		}

		resNorm = sqrtf(tempProdHost.x);
	
		if(resNorm < resNormI * solTol)
		{
			if(hipMemcpyPeerAsync(extSrcVec, extDevNum, aSrcVecDMR, devListDMR[0], vecSizeDMR, blasStreamDMR[0]) != hipSuccess)
			{
				fprintf(stderr, "solverDMR CUDA Error: Failed to copy initial guess into approximation.\n");
				fullResetDMR(numDevsDMR);
				return resNormI;
			}

			fprintf(stdout, "Initial guess within solution tolerance.\n");
			return resNorm;
		}
		// Normalize residual as first vector for Arnoldi algorithm. 
		if(hipSetDevice(devListDMR[0]) != hipSuccess)
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to set device to head inverse solver.\n");
			fullResetDMR(numDevsDMR);
			return resNormI;
		}

		normalizeVecDMR<<<blocksDMR, threadsPerBlockDMR, 0, blasStreamDMR[0]>>>(-1, numElementsDMR, resVecDMR, innProdDevDMR, 0.0);

		if(hipStreamSynchronize(blasStreamDMR[0]) != hipSuccess) 
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to synchronize head BLAS stream in source construction.\n");
			fullResetDMR(numDevsDMR);
			return resNormI;
		}
		// Iterate Arnoldi method
		while(resNorm > solTol * resNormI)
		{
			if(grdItr > matRank)
			{
				fprintf (stderr, "solverDMR Inverse Solve Error: GRD has failed to converge to a solution after %d iterations.\n", grdItr);
				numIts[0] = grdItr;
				return resNorm / resNormI;
			}

			if(resNorm / resNormI > 1.0)
			{
				fprintf (stderr, "solverDMR Inverse Solve Error: GRD Runaway!\n");
				fprintf (stderr, "\n");				
				numIts[0] = grdItr;
				return resNorm / resNormI;
			}

			if(grdItr == 0 && startDimDef == 0)
			{
				dimDeflat = 0;	
			}
			else
			{
				dimDeflat = deflateDimDMR;
			}
			
			if(arnoldiSetupDMR(dimDeflat, matRank, extSrcVec, extStream, extDevNum) != 0)
			{
				fprintf(stderr, "solverDMR Error: arnoldiSetupDMR exit.\n");
				return resNormI;
			}
			// Construct Krylov Space.
			if(arnoldiDMR(dimDeflat, extSrcVec, extStream, extDevNum, solTol) != 0)
			{
				return resNormI;
			}		 
				
			resNorm = updateAppResDMR(extSrcVec, extStream, extDevNum, dimDeflat, solTol, resNorm);

			if(localDefDimDMR > 0)
			{
				if(grdItr == 0 && startDimDef == 0)
				{
					if(genDeflatDMR(0) != 0)
					{
						fprintf(stderr, "solverDMR Error: genDeflatDMR exit.\n");
						return resNormI;
					}
				}
				else
				{
					if(genDeflatDMR(1) != 0)
					{
						fprintf(stderr, "solverDMR Error: genDeflatDMR exit.\n");
						return resNormI;
					}
				}
			}

			grdItr++;
		}
		// Export number of iterations.
		numIts[0] = grdItr - 1;

		if(hipSetDevice(extDevNum) != hipSuccess)
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to set device to global linear operator.\n");
			fullResetDMR(numDevsDMR);
			return resNormI;
		}

		if(hipMemcpyPeerAsync(extSrcVec, extDevNum, aSrcVecDMR, devListDMR[0], vecSizeDMR, extStream) != hipSuccess)
		{
			fprintf (stderr, "solverDMR CUDA Error: Failed to export solution approximation.\n");
			fullResetDMR(numDevsDMR);
			return resNormI;
		}

		if(hipStreamSynchronize(extStream) != hipSuccess) 
		{
			fprintf(stderr, "solverDMR CUDA Error: Failed to synchronize global linear operator stream.\n");
			fullResetDMR(numDevsDMR);
			return resNormI;
		}

		return resNorm / resNormI;
	}
}